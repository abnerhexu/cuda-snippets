#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "../utils/timer.h"
#include "../utils/data.h"
#include <assert.h>

// blockDim.x = 16, blockDim.y = 16
template<int BM, int BN, int BK, int TM, int TN> // TM = TN = 8, BK = 8; BM = blockDim.x*TM = 128, BN = blockDim.y*TN = 128
__global__ void tiled_mma(float *a, float *b, float *c, int M, int N, int K) {
    __shared__ float tiled_a[BM * BK]; // 128 * 8 
    __shared__ float tiled_b[BN * BK]; // 8 * 128, but transposed
    float tmp[TM * TN] = {0.0f}; // finally, compute a 4 * 4 block **every thread**
    int tid = threadIdx.x + threadIdx.y * blockDim.x; // threaed id in a block, assert it >= 128*8
    int tiled_ax = tid % 2;
    int tiled_ay = tid / 2;
    int tiled_bx = tid % 32;
    int tiled_by = tid / 32;
    int tiled_aidx = tiled_ay * BK + tiled_ax * 4;
    int tiled_bidx = tiled_bx * BK * 4 + tiled_by;
    int ay = TM * blockIdx.y * blockDim.y;
    int bx = TN * blockIdx.x * blockDim.x;
    float pb[4];
    for (int it = 0; it < K; it += BK) {
        // tiled_a[tiled_aidx] = a[(ay + tiled_ay) * K + it + tiled_ax];
        // tiled_b[tiled_bidx] = b[(tiled_by + it) * N + bx + tiled_bx];
        (float4 &)tiled_a[tiled_aidx] = (float4 &)a[(ay + tiled_ay) * K + it + 4 * tiled_ax];
        (float4 &)pb[0] = (float4 &)b[(tiled_by + it) * N + bx + 4 * tiled_bx];
        tiled_b[tiled_bidx         ] = pb[0];
        tiled_b[tiled_bidx + BK    ] = pb[1];
        tiled_b[tiled_bidx + 2 * BK] = pb[2];
        tiled_b[tiled_bidx + 3 * BK] = pb[3];
        // (float4 &)tiled_b[tiled_bidx] = (float4 &)b[(tiled_by + it) * N + bx + 4 * tiled_bx];
        __syncthreads();
        for (int p = 0; p < TM; p++) {
            for (int q = 0; q < TN; q++) {
                for (int k = 0; k < BK; k++) {
                    tmp[p * TN + q] += tiled_a[(p + threadIdx.y * TM) * BK + k] * tiled_b[(q + threadIdx.x * TN) * BK + k];
                }
            }
        }
        __syncthreads();
    }
    for (int p = 0; p < TM; p++) {
        for (int q = 0; q < TN; q++) {
            c[(ay + threadIdx.y * TM + p) * N + (bx + threadIdx.x * TN + q)] += tmp[p * TN + q]; // write back to global memory
        }
    }
}

void standard_model_mma(float* a, float* b, float* c, int M, int K, int N) {
    for (int row = 0; row < M; row++) {
        for (int col = 0; col < N; col++) {
            float sum = 0.0f;
            for (int k = 0; k < K; k++) {
                sum += a[row * K + k] * b[k * N + col];
            }
            c[row * N + col] += sum;
        }
    }
}

int main() {
    int M = 1024;
    int K = 1024;
    int N = 1024;

    auto a = generate_random_array<float>(M * K, 1.0, 1.0);
    auto b = generate_random_array<float>(K * N, 1.0, 1.0);
    auto c = generate_random_array<float>(M * N, 1.0, 1.0);

    std::vector<float> cc;
    std::copy(c.begin(), c.end(), std::back_inserter(cc));

    // Call the standard model
    standard_model_mma(a.data(), b.data(), c.data(), M, K, N);

    // Initialize matrices a, b, and c
    float* d_a, * d_b, * d_c;
    hipMalloc((void**)&d_a, M * K * sizeof(float));
    hipMalloc((void**)&d_b, K * N * sizeof(float));
    hipMalloc((void**)&d_c, M * N * sizeof(float));
    hipMemcpy(d_a, a.data(), M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), K * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c, cc.data(), M * N * sizeof(float), hipMemcpyHostToDevice);

    // Call the GPU model
    dim3 blocksPerGrid(8, 8, 1);
    dim3 threadsPerBlock(16, 16, 1);
    CUDAProgTimer timer;
    timer.start();
    for (int i = 0; i < 1; i++)
    tiled_mma<128, 128, 8, 8, 8><<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, M, K, N);
    timer.stop();
    timer.info();

    // Copy the result back to the host
    hipMemcpy(cc.data(), d_c, M * N * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Free the device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    // Result check
    assert_close<float>(c.data(), cc.data(), M * N, 1e-5);
    // for (int p = 0; p < 1024; p++) {
    //     for (int q = 0; q < 1024; q++) {
    //         // std::cout << cc[p * 1024 + q] << " ";
    //         assert(cc[p * 1024 + q] == 1);
    //     }
    //     // std::cout << std::endl;
    // }
    
    return 0;
}